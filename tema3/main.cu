
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <fstream>
#include <vector>
#include <cmath>

#define DIM_BLOCK_SIZE 1024
#define DEGREE_TO_RADIANS 0.01745329252f
#define MAX_INPUT_SIZE 500000

#define DIE(assertion, call_description)                    \
do {                                                        \
    if (assertion) {                                        \
            fprintf(stderr, "(%d): ",                       \
                            __LINE__);                      \
            perror(call_description);                       \
            exit(EXIT_FAILURE);                             \
    }                                                       \
} while(0);


__global__ void calculate_population(register float *lats, 
                                     register float *longs,
                                     register int *pops, 
                                     register int *results,
                                     register float km_range, 
                                     register size_t n) {
    register float phi1, phi2, theta1, theta2, cs, distance, index_lat, index_long, used_lat, used_long;
    register int i, curr_block_index, curr_thread_block_index, curr_pop, used_pop;
    register size_t index = threadIdx.x + blockDim.x * blockIdx.x;

    __shared__ float cached_lats[DIM_BLOCK_SIZE];
    __shared__ float cached_longs[DIM_BLOCK_SIZE];
    __shared__ int cached_pops[DIM_BLOCK_SIZE];
    cached_lats[threadIdx.x] = lats[index];
    cached_longs[threadIdx.x] = longs[index];
    cached_pops[threadIdx.x] = pops[index];

    __syncthreads();

  	if (index < n) {
        index_lat = lats[index];
        index_long = longs[index];
        curr_pop = pops[index];

        for (i = index + 1; i < n; i++) {
            curr_block_index = i / blockDim.x;
            curr_thread_block_index = i % blockDim.x;
            if (blockIdx.x == curr_block_index) {
                used_lat = cached_lats[curr_thread_block_index];
                used_long = cached_longs[curr_thread_block_index];
                used_pop = cached_pops[curr_thread_block_index];
            } else {
                used_lat = lats[i];
                used_long = longs[i];
                used_pop = pops[i];
            }

            phi1 = (90.f - index_lat) * DEGREE_TO_RADIANS;
            phi2 = (90.f - used_lat) * DEGREE_TO_RADIANS;

            theta1 = index_long * DEGREE_TO_RADIANS;
            theta2 = used_long * DEGREE_TO_RADIANS;

            cs = sin(phi1) * sin(phi2) * cos(theta1 - theta2) + cos(phi1) * cos(phi2);
            if (cs > 1) {
                cs = 1;
            } else if (cs < -1) {
                cs = -1;
            }

            distance = 6371.f * acos(cs);

            if (distance < km_range) {
                atomicAdd(&results[index], used_pop);
                atomicAdd(&results[i], curr_pop);
            }
        }
    }
}

int main(int argc, char** argv) {
    DIE( argc == 1,
         "./accpop <kmrange1> <file1in> <file1out> ...");
    DIE( (argc - 1) % 3 != 0,
         "./accpop <kmrange1> <file1in> <file1out> ...");

    int* results = NULL;

    float* device_latitudes = 0;
    float* device_longitudes = 0;
    int* device_populations = 0;
    int* device_results = 0;

    unsigned char allocated = 0;

    for(int argcID = 1; argcID < argc; argcID += 3) {
        float km_range = atof(argv[argcID]);

        std::string geon;
        float lat;
        float lon;
        int pop;

        std::vector<float> lats;
        std::vector<float> longs;
        std::vector<int> pops;

        std::ifstream ifs(argv[argcID + 1]);
        std::ofstream ofs(argv[argcID + 2]);

        while(ifs >> geon >> lat >> lon >> pop) {
            lats.push_back(lat);
            longs.push_back(lon);
            pops.push_back(pop);
        }

        allocated = 1;
        int n = (int)pops.size();

        const size_t block_size = DIM_BLOCK_SIZE;
        size_t num_blocks = n / block_size;

        if (n % DIM_BLOCK_SIZE != 0) {
            num_blocks++;
        }

        if (n > MAX_INPUT_SIZE) {
            continue;
        }

        int float_num_bytes = n * sizeof(float);
        int int_num_bytes = n * sizeof(int);

        results = (int *)malloc(int_num_bytes);

        hipMalloc((void **) &device_latitudes, float_num_bytes);
	    hipMalloc((void **) &device_longitudes, float_num_bytes);
        hipMalloc((void **) &device_populations, int_num_bytes);
        hipMalloc((void **) &device_results, int_num_bytes);

        hipMemcpy(device_latitudes, lats.data(), float_num_bytes, hipMemcpyHostToDevice);
        hipMemcpy(device_longitudes, longs.data(), float_num_bytes, hipMemcpyHostToDevice);
        hipMemcpy(device_populations, pops.data(), int_num_bytes, hipMemcpyHostToDevice);
        hipMemcpy(device_results, pops.data(), int_num_bytes, hipMemcpyHostToDevice);

        calculate_population<<<num_blocks, block_size>>>(device_latitudes, device_longitudes, device_populations, device_results, km_range, n);

        hipMemcpy(results, device_results, int_num_bytes, hipMemcpyDeviceToHost);

        for (int i = 0; i < n; i++) {
            ofs << results[i] << "\n";
        }

        ifs.close();
        ofs.close();
    }

    if (allocated) {
        free(results);

        hipFree(device_latitudes);
        hipFree(device_longitudes);
        hipFree(device_populations);
        hipFree(device_results);
    }
}
